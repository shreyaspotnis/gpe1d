#include "hip/hip_runtime.h"
// CUDA implementation of the gpe1d code
// how to compile this coming up soon, I have something very specific on my
// computer
/*
Copyright 2012 Shreyas Potnis

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.

*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <hipfft/hipfft.h>
#include "pca_utils.h"

#define BLOCKSIZE 512 

// Set this during compile
#ifdef _USE_DOUBLE_PRECISION
typedef double2 Complex;
typedef double Ipp;
#else
typedef float2 Complex;
typedef float Ipp;
#endif

int readInt ( FILE *fp) {
    int a;
    fread (&a, 1, sizeof(int), fp );
    return a;
}

Ipp readFloat ( FILE *fp) {
    Ipp a;
    fprintf(stderr, "%d", sizeof(Ipp));
    fread (&a, 1, sizeof(Ipp), fp );
    return a;
}

struct InputData {
    int Nx;
    int Ntstore;
    int Ntskip;
    int imag_time;
    Ipp C1;
    Ipp dx;
};

// The same as the x unitary we had in our CPU version
static __global__ void x_unitary(int Nx, Complex *psiX, Complex *U1c, 
                                Ipp C1)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < Nx; i += numThreads) {
        Ipp an, es, ec, temp;
        an = C1*(psiX[i].x*psiX[i].x + psiX[i].y*psiX[i].y);
        es = sin(an);
        ec = cos(an);
        temp = psiX[i].x;
        psiX[i].x = ( ec * psiX[i].x - es * psiX[i].y );
        psiX[i].y = ( ec * psiX[i].y + es * temp );
        temp = psiX[i].x;
        psiX[i].x = ( U1c[i].x * psiX[i].x - U1c[i].y * psiX[i].y );
        psiX[i].y = ( U1c[i].x * psiX[i].y + U1c[i].y * temp );
    }
}

// The same as the x unitary we had in our CPU version
static __global__ void x_unitary_imag(int Nx, Complex *psiX, Complex *U1c, 
                                        Ipp C1)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < Nx; i += numThreads) {
        Ipp an, ex, temp;
        an = C1*(psiX[i].x*psiX[i].x + psiX[i].y*psiX[i].y);
        ex = exp(an);
        psiX[i].x *= ex;
        psiX[i].y *= ex;
        temp = psiX[i].x;
        psiX[i].x = ( U1c[i].x * psiX[i].x - U1c[i].y * psiX[i].y );
        psiX[i].y = ( U1c[i].x * psiX[i].y + U1c[i].y * temp );
    }
}

static __global__ void k_unitary(int Nx, Complex *psiX, Complex *Kinc)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < Nx; i += numThreads) {
        Ipp temp;
        temp = psiX[i].x;
        psiX[i].x = ( Kinc[i].x * psiX[i].x - Kinc[i].y * psiX[i].y );
        psiX[i].y = ( Kinc[i].x * psiX[i].y + Kinc[i].y * temp );
    }
}
/*
static __global__ void psi_length(int Nx, Complex *psiX, Ipp *sum_total, 
                Ipp dx)
{
    // note: works only for Nx which are powers of 2
    __shared__ Ipp  sum[BLOCKSIZE];
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    sum[threadIdx.x] = psiX[i].x*psiX[i].x + psiX[i].y*psiX[i].y;
    // To make sure all threads in a block have the sum[] value:
    __syncthreads();
    int nTotalThreads = blockDim.x;  // Total number of active threads;
    // only the first half of the threads will be active.
    while(nTotalThreads > 1) {
        int halfPoint = (nTotalThreads >> 1);	// divide by two
        if (threadIdx.x < halfPoint) {
            int thread2 = threadIdx.x + halfPoint;
            sum[threadIdx.x] += sum[thread2];  // Pairwise summation
        }
        __syncthreads();
        nTotalThreads = halfPoint;  // Reducing the binary tree size by two
    }

    if (threadIdx.x == 0) {
      atomicAdd (sum_total, sum[0]*dx);
    }

    return;
}*/

static __global__ void psi_block_length(int Nx, Complex* psiX, Ipp
                                        *psi_block_sum, Ipp dx) {
    // note: works only for Nx which are powers of 2
    __shared__ Ipp  sum[BLOCKSIZE];
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    sum[threadIdx.x] = psiX[i].x*psiX[i].x + psiX[i].y*psiX[i].y;
    // To make sure all threads in a block have the sum[] value:
    __syncthreads();
    int nTotalThreads = blockDim.x;  // Total number of active threads;
    // only the first half of the threads will be active.
    while(nTotalThreads > 1) {
        int halfPoint = (nTotalThreads >> 1);	// divide by two
        if (threadIdx.x < halfPoint) {
            int thread2 = threadIdx.x + halfPoint;
            sum[threadIdx.x] += sum[thread2];  // Pairwise summation
        }
        __syncthreads();
        nTotalThreads = halfPoint;  // Reducing the binary tree size by two
    }
    if (threadIdx.x == 0)
        psi_block_sum[blockIdx.x] = sum[0] * dx;
    return;
}

static __global__ void psi_total_length(Ipp *psi_block_sum,
                                        Ipp *psi_total_sum) {
    extern __shared__ Ipp sum[];
    // Copying from global to shared memory: 
    sum[threadIdx.x] = psi_block_sum[threadIdx.x];
    // To make sure all threads in a block have the sum[] value:
    __syncthreads();
    int nTotalThreads = blockDim.x;  // Total number of active threads;
    // only the first half of the threads will be active.
    while(nTotalThreads > 1) {
        int halfPoint = (nTotalThreads >> 1); // divide by two
        if (threadIdx.x < halfPoint) {
            int thread2 = threadIdx.x + halfPoint;
            sum[threadIdx.x] += sum[thread2];  // Pairwise summation
        }
        __syncthreads();
        nTotalThreads = halfPoint;
    }
    if (threadIdx.x == 0) {
        *psi_total_sum = sum[0];
    }
    return;
}

static __global__ void normalize_psi(int Nx, Complex *psiX, Ipp *sum_total)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < Nx; i += numThreads) {
        psiX[i].x /= sqrt(*sum_total);
        psiX[i].y /= sqrt(*sum_total);
    }
}

int main()
{
    fprintf(stderr, "Start program\n");
    InputData input;
    fread (&input, 1, sizeof(InputData), stdin);
    int Nx = input.Nx;
    int Ntstore = input.Ntstore;
    int Ntskip = input.Ntskip;
    int imag_time = input.imag_time;
    Ipp C1 = input.C1;
    Ipp dx = input.dx;

    /*
    int Nx = readInt(stdin);
    int Ntstore = readInt(stdin);
    int Ntskip = readInt(stdin);
    Ipp C1 = readFloat(stdin);
    Ipp dx = readFloat(stdin);
    int imag_time = readInt(stdin);*/
    fprintf(stderr, "have ints and doubles\n");
    fprintf(stderr, "Nx:%d Ntstore %d Ntskip %d C1 %f dx %f imag_time %d",
                    Nx, Ntstore, Ntskip, C1, dx, imag_time);
    
    int memSize;
    int blockSize, nBlocks;
    Complex *psiX, *U1c, *Kinc;
    Complex *psiX_d, *U1c_d, *Kinc_d;
    Ipp *psi_sum_d;
    Ipp *psi_block_sum_d;

    // allocate memory
    memSize = sizeof(Complex) * Nx;

    psiX = (Complex*)malloc(memSize);
    U1c = (Complex*)malloc(memSize);
    Kinc = (Complex*)malloc(memSize);

    // allocate memory on the device
    hipMalloc((void**)&psiX_d, memSize);
    hipMalloc((void**)&U1c_d, memSize);
    hipMalloc((void**)&Kinc_d, memSize);
    hipMalloc((void**)&psi_sum_d, sizeof(Ipp));

    fprintf(stderr, "size of psiX cuda:%d\n", Nx * sizeof(Complex));
    fread(psiX, Nx, sizeof(Complex), stdin);
    fprintf(stderr, "have psiX");

    fread(U1c, Nx, sizeof(Complex), stdin);
    fprintf(stderr, "have U1c");
    fread(Kinc, Nx, sizeof(Complex), stdin);
    fprintf(stderr, "have Kinc");

    fprintf(stderr, "have arrays\n");
    // copy data to the device
    hipMemcpy(psiX_d, psiX , memSize, hipMemcpyHostToDevice);
    hipMemcpy(U1c_d, U1c, memSize, hipMemcpyHostToDevice);
    hipMemcpy(Kinc_d, Kinc , memSize, hipMemcpyHostToDevice);

    // CUFFT plan
    hipfftHandle plan;
#ifdef _USE_DOUBLE_PRECISION
    hipfftPlan1d(&plan, Nx, HIPFFT_Z2Z, 1);
#else
    hipfftPlan1d(&plan, Nx, HIPFFT_C2C, 1);
#endif

    /* set up device execution configuration */
    blockSize = BLOCKSIZE;
    nBlocks = Nx / blockSize + (Nx % blockSize > 0);
    hipMalloc((void**)&psi_block_sum_d, nBlocks * sizeof(Ipp));
    // initialize block sum to zero
    hipMemset(psi_block_sum_d, 0, nBlocks * sizeof(Ipp));
    
    pca_time tt;
    tick(&tt);
    if(!imag_time) {
            fwrite(psiX, Nx, sizeof(Complex), stdout);
    }
    for(int t1=0; t1<Ntstore-1; t1++) {
        for(int t2=0; t2<Ntskip; t2++) {
            if(imag_time)
                x_unitary_imag<<<nBlocks, blockSize>>>(Nx, psiX_d, U1c_d, C1);
            else
                x_unitary<<<nBlocks, blockSize>>>(Nx, psiX_d, U1c_d, C1);
            hipfftExecC2C(plan, (hipfftComplex *)psiX_d,
                            (hipfftComplex *)psiX_d, HIPFFT_FORWARD);
            k_unitary<<<nBlocks, blockSize>>>(Nx, psiX_d, Kinc_d);
            hipfftExecC2C(plan, (hipfftComplex *)psiX_d,
                            (hipfftComplex *)psiX_d, HIPFFT_BACKWARD);
            if(imag_time)
                x_unitary_imag<<<nBlocks, blockSize>>>(Nx, psiX_d, U1c_d, C1);
            else
                x_unitary<<<nBlocks, blockSize>>>(Nx, psiX_d, U1c_d, C1);
            hipDeviceSynchronize ();

            if(imag_time) {
                // psi_length<<<nBlocks, blockSize>>>(Nx, psiX_d, psi_sum_d, dx);
                psi_block_length<<<nBlocks, blockSize>>>(Nx, psiX_d,
                                                         psi_block_sum_d, dx); 
                psi_total_length<<<1, nBlocks, nBlocks * sizeof(Ipp)>>>
                                                (psi_block_sum_d, psi_sum_d);

                normalize_psi<<<nBlocks, blockSize>>>(Nx, psiX_d, psi_sum_d);
             }               
        }
        if(!imag_time) {
            // send the output to stdout, our main process will catch it
            hipMemcpy(psiX, psiX_d, memSize, hipMemcpyDeviceToHost);
            fwrite(psiX, Nx, sizeof(Complex), stdout);
        }
    }
    if(imag_time) {
        hipMemcpy(psiX, psiX_d, memSize, hipMemcpyDeviceToHost);
        fwrite(psiX, Nx, sizeof(Complex), stdout);
    }

    tock(&tt);

    // release memory 
    hipFree(psiX_d);
    hipFree(U1c_d);
    hipFree(Kinc_d);
    hipFree(psi_sum_d);
    hipFree(psi_block_sum_d);
    free(psiX);
    free(U1c);
    free(Kinc);
    return 0;
}
