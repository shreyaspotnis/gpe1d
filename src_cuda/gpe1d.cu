#include "hip/hip_runtime.h"
// CUDA implementation of the gpe1d code
// how to compile this coming up soon, I have something very specific on my
// computer
/*
Copyright 2012 Shreyas Potnis

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.

*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <hipfft/hipfft.h>
#include "pca_utils.h"

#define BLOCKSIZE 512 

typedef float2 Complex; 
__device__ float psi_sum_d;

int readInt ( FILE *fp) {
    int a;
    fread (&a, 1, sizeof(int), fp );
    return a;
}

float readFloat ( FILE *fp) {
    float a;
    fread (&a, 1, sizeof(float), fp );
    return a;
}

// The same as the x unitary we had in our CPU version
static __global__ void x_unitary(int Nx, Complex *psiX, Complex *U1c, 
                                float C1)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < Nx; i += numThreads) {
        float an, es, ec, temp;
        an = C1*(psiX[i].x*psiX[i].x + psiX[i].y*psiX[i].y);
        es = sin(an);
        ec = cos(an);
        temp = psiX[i].x;
        psiX[i].x = ( ec * psiX[i].x - es * psiX[i].y );
        psiX[i].y = ( ec * psiX[i].y + es * temp );
        temp = psiX[i].x;
        psiX[i].x = ( U1c[i].x * psiX[i].x - U1c[i].y * psiX[i].y );
        psiX[i].y = ( U1c[i].x * psiX[i].y + U1c[i].y * temp );
    }
}

// The same as the x unitary we had in our CPU version
static __global__ void x_unitary_imag(int Nx, Complex *psiX, Complex *U1c, 
                                        float C1)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < Nx; i += numThreads) {
        float an, ex, temp;
        an = C1*(psiX[i].x*psiX[i].x + psiX[i].y*psiX[i].y);
        ex = exp(an);
        psiX[i].x *= ex;
        psiX[i].y *= ex;
        temp = psiX[i].x;
        psiX[i].x = ( U1c[i].x * psiX[i].x - U1c[i].y * psiX[i].y );
        psiX[i].y = ( U1c[i].x * psiX[i].y + U1c[i].y * temp );
    }
}

static __global__ void k_unitary(int Nx, Complex *psiX, Complex *Kinc)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < Nx; i += numThreads) {
        float temp;
        temp = psiX[i].x;
        psiX[i].x = ( Kinc[i].x * psiX[i].x - Kinc[i].y * psiX[i].y );
        psiX[i].y = ( Kinc[i].x * psiX[i].y + Kinc[i].y * temp );
    }
}

static __global__ void psi_length(int Nx, Complex *psiX, float *sum_total, 
                float dx)
{
    // note: works only for Nx which are powers of 2
    __shared__ float  sum[BLOCKSIZE];
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    sum[threadIdx.x] = psiX[i].x*psiX[i].x + psiX[i].y*psiX[i].y;
    // To make sure all threads in a block have the sum[] value:
    __syncthreads();
    int nTotalThreads = blockDim.x;  // Total number of active threads;
    // only the first half of the threads will be active.
    while(nTotalThreads > 1) {
        int halfPoint = (nTotalThreads >> 1);	// divide by two
        if (threadIdx.x < halfPoint) {
            int thread2 = threadIdx.x + halfPoint;
            sum[threadIdx.x] += sum[thread2];  // Pairwise summation
        }
        __syncthreads();
        nTotalThreads = halfPoint;  // Reducing the binary tree size by two
    }

    if (threadIdx.x == 0) {
      atomicAdd (sum_total, sum[0]*dx);
    }

    return;
}

static __global__ void normalize_psi(int Nx, Complex *psiX, float *sum_total)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < Nx; i += numThreads) {
        psiX[i].x /= sqrt(*sum_total);
        psiX[i].y /= sqrt(*sum_total);
    }
}

int main()
{
    int Nx = readInt(stdin);
    int Ntstore = readInt(stdin);
    int Ntskip = readInt(stdin);
    float C1 = readFloat(stdin);
    float dx = readFloat(stdin);
    int imag_time = readInt(stdin);
    
    int memSize;
    int blockSize, nBlocks;
    Complex *psiX, *U1c, *Kinc;
    Complex *psiX_d, *U1c_d, *Kinc_d;
    float *psi_sum_d;

    // allocate memory
    memSize = sizeof(Complex) * Nx;

    psiX = (Complex*)malloc(memSize);
    U1c = (Complex*)malloc(memSize);
    Kinc = (Complex*)malloc(memSize);

    // allocate memory on the device
    hipMalloc((void**)&psiX_d, memSize);
    hipMalloc((void**)&U1c_d, memSize);
    hipMalloc((void**)&Kinc_d, memSize);
    hipMalloc((void**)&psi_sum_d, sizeof(float));

    fread(psiX, Nx, sizeof(Complex), stdin);
    fread(U1c, Nx, sizeof(Complex), stdin);
    fread(Kinc, Nx, sizeof(Complex), stdin);

    // copy data to the device
    hipMemcpy(psiX_d, psiX , memSize, hipMemcpyHostToDevice);
    hipMemcpy(U1c_d, U1c, memSize, hipMemcpyHostToDevice);
    hipMemcpy(Kinc_d, Kinc , memSize, hipMemcpyHostToDevice);

    // CUFFT plan
    hipfftHandle plan;
    hipfftPlan1d(&plan, Nx, HIPFFT_C2C, 1);

    /* set up device execution configuration */
    blockSize = BLOCKSIZE;
    nBlocks = Nx / blockSize + (Nx % blockSize > 0);
    
    pca_time tt;
    tick(&tt);
    if(!imag_time) {
            fwrite(psiX, Nx, sizeof(Complex), stdout);
    }
    for(int t1=0; t1<Ntstore-1; t1++) {
        for(int t2=0; t2<Ntskip; t2++) {
            if(imag_time)
                x_unitary_imag<<<nBlocks, blockSize>>>(Nx, psiX_d, U1c_d, C1);
            else
                x_unitary<<<nBlocks, blockSize>>>(Nx, psiX_d, U1c_d, C1);
            hipfftExecC2C(plan, (hipfftComplex *)psiX_d,
                            (hipfftComplex *)psiX_d, HIPFFT_FORWARD);
            k_unitary<<<nBlocks, blockSize>>>(Nx, psiX_d, Kinc_d);
            hipfftExecC2C(plan, (hipfftComplex *)psiX_d,
                            (hipfftComplex *)psiX_d, HIPFFT_BACKWARD);
            if(imag_time)
                x_unitary_imag<<<nBlocks, blockSize>>>(Nx, psiX_d, U1c_d, C1);
            else
                x_unitary<<<nBlocks, blockSize>>>(Nx, psiX_d, U1c_d, C1);
            hipDeviceSynchronize ();

            if(imag_time) {
                float zero_float = 0.0;
                hipMemcpy(psi_sum_d, &zero_float , sizeof(float),
                              hipMemcpyHostToDevice);
                psi_length<<<nBlocks, blockSize>>>(Nx, psiX_d, psi_sum_d, dx);
                normalize_psi<<<nBlocks, blockSize>>>(Nx, psiX_d, psi_sum_d);
             }               
        }
        if(!imag_time) {
            // send the output to stdout, our main process will catch it
            hipMemcpy(psiX, psiX_d, memSize, hipMemcpyDeviceToHost);
            fwrite(psiX, Nx, sizeof(Complex), stdout);
        }
    }
    if(imag_time) {
        hipMemcpy(psiX, psiX_d, memSize, hipMemcpyDeviceToHost);
        fwrite(psiX, Nx, sizeof(Complex), stdout);
    }

    tock(&tt);

    // release memory 
    hipFree(psiX_d);
    hipFree(U1c_d);
    hipFree(Kinc_d);
    hipFree(psi_sum_d);
    free(psiX);
    free(U1c);
    free(Kinc);
    return 0;
}
